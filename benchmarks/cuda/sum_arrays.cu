
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__
void map2_xy_kernel(float *a1, float *a2, float *r, int size, float (*f)(float*,float*,int))
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
if((id < size))
{
        r[id] = f(a1, a2, id);
}

}

__device__
float anonymous(float *a1, float *a2, int i)
{
return ((a1[i] + a2[i]));
}

__device__ void* anonymous_ptr = (void*) anonymous;

extern "C" void* get_anonymous_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_ptr), sizeof(void*));
        return host_function_ptr;
}





int main(int argc, char *argv[])
{

    float *a, *b, *resp;
	float *dev_a, *dev_b, *dev_resp;
    hipError_t j_error;

    int N = atoi(argv[1]);

    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    resp = (float*)malloc(N*sizeof(float));

    for(int i=0; i<N; i++) {
		a[i] = 1;
		
	}

    for(int i=0; i<N; i++) {
		b[i] = 2;
		
	}


    int threadsPerBlock = 128;
    int  numberOfBlocks = (N + threadsPerBlock - 1)/ threadsPerBlock;

    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


	hipMalloc((void**)&dev_a, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMalloc((void**)&dev_b, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMalloc((void**)&dev_resp, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    float (*f)(float*,float*,int) = (float (*)(float*,float*,int)) get_anonymous_ptr();

   map2_xy_kernel<<< numberOfBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_resp, N, f);
    
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    hipMemcpy(resp, dev_resp, N*sizeof(float), hipMemcpyDeviceToHost);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    
    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_resp);
    
	hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", N,time);

/*
    for(int i=0; i<10; i++) {
		printf("resp[%d] = %f;\n",i,resp[i]);
	}
*/

	//printf("\n FINAL RESULTADO: %f \n", c);

	free(a);
    free(b);
	free(resp);

}
