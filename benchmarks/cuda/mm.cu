
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
__global__ void gpu_mm(float *a,float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

void cpu_mm(float *h_a, float *h_b, float *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

void checkElementsAre(float *gpu, float *cpu, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(gpu[i] != cpu[i])
    {
      printf("FAIL: gpu[%d] - %0.0f does not equal cpu = %0.0f\n", i, gpu[i], cpu[i]);
      exit(1);
    }
  }
  printf("SUCCESS! All values computed correctly.\n");
}

int main(int argc, char const *argv[])
{   
    
    int value = atoi(argv[1]);
    
    
    int m = value;
    int block_size = 16;
    hipError_t j_error;
    

    float *a = (float*) malloc(m*m*sizeof(float));
    float *b = (float*) malloc(m*m*sizeof(float));
    float *c = (float*) malloc(m*m*sizeof(float));
    float *cpu_result = (float*) malloc(m*m*sizeof(float));
    
    srand(time(0));

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            a[i * m + j] =  (rand() %(100 -1 + 1)) + 1;
        }
    }

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            b[i * m + j] = (rand() %(100 -1 + 1)) + 1;
        }
    }


    //for (int i=0;i<m;i++)
    //    printf("v %f\n",b[10]);
    float *d_a, *d_b, *d_c;

    int grid_rows = (m + block_size - 1) / block_size;
    int grid_cols = (m + block_size - 1) / block_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);
   
   //struct timespec begin, end;
   //clock_gettime(CLOCK_MONOTONIC, &begin);

   float time;
    hipEvent_t start, stop;   
     hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    
    hipMalloc((void **) &d_a, sizeof(float)*m*m);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(j_error));
    hipMalloc((void **) &d_b, sizeof(float)*m*m);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(j_error));
    hipMalloc((void **) &d_c, sizeof(float)*m*m);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(j_error));
   
   
    hipMemcpy(d_a, a, sizeof(float)*m*m, hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(j_error));
    hipMemcpy(d_b, b, sizeof(float)*m*m, hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 5: %s\n", hipGetErrorString(j_error));
    
    
    gpu_mm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m,m,m);  
   
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 6: %s\n", hipGetErrorString(j_error));
    
    hipDeviceSynchronize();
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Synchronize: %s\n", hipGetErrorString(j_error));

    hipMemcpy(c, d_c, sizeof(float)*m*m, hipMemcpyDeviceToHost);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(j_error));

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("cuda\t%d\t%3.1f\n", m,time);

    //clock_gettime(CLOCK_MONOTONIC_RAW, &end);

    //printf ("cuda   %d   %f \n",m,
     //       ((end.tv_nsec - begin.tv_nsec) / 1000000000.0 +
       //     (end.tv_sec  - begin.tv_sec))*1000);
   
//    cpu_mm(a,b,cpu_result,m,m,m);
  
  //  checkElementsAre(c,cpu_result,m*m);

    
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
    