#include<stdio.h>


#include <hip/hip_runtime.h>
#include <builtin_types.h>


__global__
void inc2(float v)
{
 int x = v;
}


__device__
float inc(float v)
{
 return v+2;
}

__global__
void inc_vet(float *result, float *a, int n)
{
	// void **fun_res;

	float (*fun)(float) = inc; 
	int i= (threadIdx.x + (blockIdx.x * blockDim.x));
	if(i < n)   
            result[i] = fun(a[i]);
}




int main (int argc, char *argv[]) {
	float *a, *resp, *dev_a, *dev_resp;

	int n = 10000;

	int block_size = 32;
	int nBlocks = (n + block_size - 1) / block_size;

	printf("block_size = %d   nBlocks = %d total = %d\n", block_size,nBlocks,block_size*nBlocks);

	a = (float*)malloc(n*sizeof(float));

	
	resp = (float*)malloc(n*sizeof(float));

	for(int i=0; i<n; i++) {
		a[i] = i;
		
	}
    
	hipMalloc((void**)&dev_a, n*sizeof(float));
	hipMalloc((void**)&dev_resp, n*sizeof(float));
	
    hipMemcpy(dev_a, a, n*sizeof(float), hipMemcpyHostToDevice);


	hipModule_t cuModule;

    int ret = hipModuleLoad(&cuModule, "/home/dubois/hok/loadModule.ptx");

	printf("retorno %d\n", ret);

	hipFunction_t function;
    int funs = 0;
//	ret = cuModuleGetFunctionCount(&funs,cuModule) ;

	printf("retorno %d funs= %d\n", ret,funs);


	ret = hipModuleGetFunction(&function, cuModule, "inc2");

	printf("retorno %d\n", ret);



	inc_vet<<<nBlocks, block_size>>>(dev_resp, dev_a , n);

	hipMemcpy(resp,dev_resp, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<10; i++) {
		printf("resp[%i] = %f\n", i,resp[i]);
	}
   
	hipFree(dev_a);
	hipFree(dev_resp);
    
	
	free(a);
	free(resp);
  	
}