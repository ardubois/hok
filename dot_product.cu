#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#define imin(a,b) (a<b?a:b)





__global__ void dot(float* a, float* b, float* c, int N) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < N){
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x/2;
	while (i != 0){
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];

		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main (int argc, char *argv[]) {
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;
	
	int N = atoi(argv[1]);
   
const int threadsPerBlock = 256;

	int blocksPerGrid =  (N+threadsPerBlock-1) / threadsPerBlock;

	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));
	
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	for(int i=0; i<N; i++) {
		a[i] = 1;
		b[i] = 1;
	}

	float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


	hipMalloc((void**)&dev_a, N*sizeof(float));
	hipMalloc((void**)&dev_b, N*sizeof(float));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float));
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

	dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c, N);

	hipMemcpy(partial_c,dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

	c = 0;
	for(int i=0; i<blocksPerGrid; i++) {
		printf("vet %d = %f\n",i,partial_c[i]);
		c += partial_c[i];
	}
    printf("result %d %d", c, blocksPerGrid);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
    
	hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", N,time);

	//printf("\n FINAL RESULTADO: %f \n", c);

	free(a);
	free(b);
	free(partial_c);
  	
}