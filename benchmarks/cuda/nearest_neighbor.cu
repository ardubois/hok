/*
 * nn.cu
 * Nearest Neighbor
 * Modified by André Du Bois: changed depracated api, creating data set in memory. clean up code not used
 */

#include <stdio.h>
#include <sys/time.h>
#include <float.h>
#include <vector>
#include "hip/hip_runtime.h"

#include <time.h>


__device__ static float atomic_cas(float* address, float oldv, float newv)
{
    int* address_as_i = (int*) address;
    return  __int_as_float(atomicCAS(address_as_i, __float_as_int(oldv), __float_as_int(newv)));
}



__global__
void reduce_kernel(float *a, float *ref4, float (*f)(float,float), int n)
{
__shared__ float cache[256];
        int tid = (threadIdx.x + (blockIdx.x * blockDim.x));
        int cacheIndex = threadIdx.x;
        float temp = 0.0;
if((tid < n))
{
        temp = a[tid];
        tid = ((blockDim.x * gridDim.x) + tid);
}

while((tid < n)){
        temp = f(a[tid], temp);
        tid = ((blockDim.x * gridDim.x) + tid);
}
        cache[cacheIndex] = temp;
__syncthreads();
        int i = (blockDim.x / 2);
        int up = ((blockDim.x * gridDim.x) * 256);
while(((i != 0) && ((cacheIndex + up) < n))){
if((cacheIndex < i))
{
        cache[cacheIndex] = f(cache[(cacheIndex + i)], cache[cacheIndex]);
}

__syncthreads();

        i = (i / 2);
}
if((cacheIndex == 0))
{
        float current_value = ref4[0];
while((! (current_value == atomic_cas(ref4, current_value, f(cache[0], current_value))))){
        current_value = ref4[0];
}
}

}
__global__
void map_step_2para_1resp_kernel(float *d_array, float *d_result, int step, float par1, float par2, int size, float (*f)(float*,float,float))
{
int globalId = ((blockDim.x * ((gridDim.x * blockIdx.y) + blockIdx.x)) + threadIdx.x);
int id = (step * globalId);
if((globalId < size))
{
        d_result[globalId] = f((d_array + id), par1, par2);
}

}

__device__
float euclid(float *d_locations, float lat, float lng)
{
return (sqrt((((lat - d_locations[0]) * (lat - d_locations[0])) + ((lng - d_locations[1]) * (lng - d_locations[1])))));
}

__device__ void* euclid_ptr = (void*) euclid;

extern "C" void* get_euclid_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(euclid_ptr), sizeof(void*));
        return host_function_ptr;
}

__device__
float menor(float x, float y)
{
if((y == 0.0))
{
return (x);
}
else{
if((x < y))
{
return (x);
}
else{
return (y);
}

}

}

__device__ void* menor_ptr = (void*) menor;

extern "C" void* get_menor_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(menor_ptr), sizeof(void*));
        return host_function_ptr;
}



/*
typedef struct latLong
{
  float lat;
  float lng;
} LatLong;

typedef struct record
{
  char recString[REC_LENGTH];
  float distance;
} Record;
*/
void loadData(float *locations, int size);
//void findLowest(std::vector<Record> &records,float *distances,int numRecords,int topN);
//void printUsage();
//int parseCommandline(int argc, char *argv[], char* filename,int *r,float *lat,float *lng,
//                     int *q, int *t, int *p, int *d);

/**
* Kernel
* Executed on GPU
* Calculates the Euclidean distance from each record in the database to the target position

__global__ void euclid(LatLong *d_locations, float *d_distances, int numRecords,float lat, float lng)
{
	//int globalId = gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
	int globalId = blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x ) + threadIdx.x; // more efficient
    LatLong *latLong = d_locations+globalId;
    if (globalId < numRecords) {
        float *dist=d_distances+globalId;
        *dist = (float)sqrt((lat-latLong->lat)*(lat-latLong->lat)+(lng-latLong->lng)*(lng-latLong->lng));
	}
}
**/
/**
* This program finds the k-nearest neighbors
**/

int main(int argc, char* argv[])
{
//	int    i=0;
	//float lat=0, lng=0;
	
  //  std::vector<Record> records;
	float *locations;

  int numRecords = atoi(argv[1]);
    
   locations = (float *)malloc(sizeof(float) * 2*numRecords);
   // int numRecords = loadData(filename,records,locations);
   loadData(locations,numRecords);

    
	float *distances;
	//Pointers to device memory
	float *d_locations;
	float *d_distances;


	

	/**
	
	* Allocate memory on host and device

  */

  float time;
    hipEvent_t start, stop;   
     hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    //int size_dist = numRecords/2;
	distances = (float *)malloc(sizeof(float) * numRecords);
	hipMalloc((void **) &d_locations,sizeof(float) * 2 * numRecords);
	hipMalloc((void **) &d_distances,sizeof(float) * numRecords);

   /**
    * Transfer data from host to device
    */
    hipMemcpy( d_locations, &locations[0], sizeof(float) * 2* numRecords, hipMemcpyHostToDevice);

    /**
    * Execute kernel --
    */

   


    float (*f1)(float*,float,float) = (float (*)(float*,float,float)) get_euclid_ptr();
    float (*f2)(float,float) = (float (*)(float,float)) get_menor_ptr();

    map_step_2para_1resp_kernel<<< numRecords, 1 >>>(d_locations,d_distances,2,0.0,0.0, numRecords, f1);
    

    hipDeviceSynchronize();

    int threadsPerBlock = 256;
    int blocksPerGrid = (numRecords + threadsPerBlock - 1)/ threadsPerBlock;

    float *resp, *d_resp;
    resp = (float *)malloc(sizeof(float));
	hipMalloc((void **) &d_resp,sizeof(float));


    reduce_kernel<<< blocksPerGrid, threadsPerBlock >>>(d_distances,d_resp,f2,numRecords);
    hipDeviceSynchronize();
    //Copy data from device memory to host memory

    hipMemcpy( resp, d_resp, sizeof(float), hipMemcpyDeviceToHost );


	// find the resultsCount least distances
    free(distances);
    //Free memory
	hipFree(d_locations);
	hipFree(d_distances);
   
     hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

     printf("CUDA\t%d\t%3.1f\n", numRecords,time);

}

void loadData(float* locations, int size){
   
	for (int i=0;i<size;i++){
			
            locations[0] = ((float)(7 + rand() % 63)) + ((float) rand() / (float) 0x7fffffff);

            locations[1] = ((float)(rand() % 358)) + ((float) rand() / (float) 0x7fffffff); 

            locations = locations +2;
            
           
        }
     
}



