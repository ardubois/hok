
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__device__
float saxpy(float a, float b)
{
return (((2 * a) + b));
}

__device__ void* saxpy_ptr = (void*) saxpy;

extern "C" void* get_saxpy_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(saxpy_ptr), sizeof(void*));
        return host_function_ptr;
}

__global__
void map_2kernel(float *a1, float *a2, float *a3, int size, float (*f)(float,float))
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
int stride = (blockDim.x * gridDim.x);
for( int i = id; i<size; i+=stride){
if((id < size))
{
        a3[id] = f(a1[id], a2[id]);
}

}

}


int main(int argc, char *argv[])
{

    float *a, *b, *resp;
	float *dev_a, *dev_b, *dev_resp;
    hipError_t j_error;

    int N = atoi(argv[1]);

    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    resp = (float*)malloc(N*sizeof(float));

    for(int i=0; i<N; i++) {
		a[i] = rand();
		
	}

    for(int i=0; i<N; i++) {
		b[i] =rand();
		
	}


    //int threadsPerBlock = 128;
    //int  numberOfBlocks = (N + threadsPerBlock - 1)/ threadsPerBlock;

   int threadsPerBlock = 256;
   int   numberOfBlocks = 1024;

    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


	hipMalloc((void**)&dev_a, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMalloc((void**)&dev_b, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMalloc((void**)&dev_resp, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    float (*f)(float,float) = (float (*)(float,float)) get_saxpy_ptr();

    map_2kernel<<< numberOfBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_resp, N, f);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    hipMemcpy(resp, dev_resp, N*sizeof(float), hipMemcpyDeviceToHost);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    
    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_resp);
    
	hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", N,time);

/*
    for(int i=0; i<10; i++) {
		printf("resp[%d] = %f;\n",i,resp[i]);
	}

*/
	//printf("\n FINAL RESULTADO: %f \n", c);

	free(a);
    free(b);
	free(resp);

}
