
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


__global__
void map2xy2D_kernel(float *arr1, float *arr2, int par, float *resp, int size, float (*f)(float*,float*,int,int,int))
{
        int row = ((blockIdx.y * blockDim.y) + threadIdx.y);
        int col = ((blockIdx.x * blockDim.x) + threadIdx.x);



if(((col < size) && (row < size)))
{
        resp[((row * size) + col)] = f(arr1, arr2, par, row, col);
}

}



__device__
float anonymous_9nl89mhko6(float *mat1, float *mat2, int m, int x, int y)
{
        float sum = 0.0;
for( int i = 0; i<m; i+=1){
        sum = (sum + (mat1[((x * m) + i)] * mat2[((i * m) + y)]));
}

return (sum);
}

__device__ void* anonymous_9nl89mhko6_ptr = (void*) anonymous_9nl89mhko6;

void* get_anonymous_9nl89mhko6_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_9nl89mhko6_ptr), sizeof(void*));
        return host_function_ptr;
}







void cpu_mm(float *h_a, float *h_b, float *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

void checkElementsAre(float *gpu, float *cpu, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(gpu[i] != cpu[i])
    {
      printf("FAIL: gpu[%d] - %0.0f does not equal cpu = %0.0f\n", i, gpu[i], cpu[i]);
      exit(1);
    }
  }
  printf("SUCCESS! All values computed correctly.\n");
}

int main(int argc, char const *argv[])
{   
    
    int value = atoi(argv[1]);
    
    
    int m = value;
    
    hipError_t j_error;
    

    float *a = (float*) malloc(m*m*sizeof(float));
    float *b = (float*) malloc(m*m*sizeof(float));
    float *c = (float*) malloc(m*m*sizeof(float));
    float *cpu_result = (float*) malloc(m*m*sizeof(float));
    
    srand(time(0));
    /*
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            a[i * m + j] =  (rand() %(100 -1 + 1)) + 1;
        }
    }

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            b[i * m + j] = (rand() %(100 -1 + 1)) + 1;
        }
    }
    */

for (int i = 1; i <= m*m; ++i) {
    a[i] = rand() %1000;
}


for (int i = 1; i <= m*m; ++i) {
    b[i] = rand() %1000;
}

  


    //for (int i=0;i<m;i++)
    //    printf("v %f\n",b[10]);
    float *d_a, *d_b, *d_c;

  int block_size = 16;
    int grid_rows = (m + block_size - 1) / block_size;
    int grid_cols = (m + block_size - 1) / block_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);
   
   //struct timespec begin, end;
   //clock_gettime(CLOCK_MONOTONIC, &begin);

   float time;
    hipEvent_t start, stop;   
     hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    
    hipMalloc((void **) &d_a, sizeof(float)*m*m);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(j_error));
    hipMalloc((void **) &d_b, sizeof(float)*m*m);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(j_error));
    hipMalloc((void **) &d_c, sizeof(float)*m*m);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(j_error));
   
   
    hipMemcpy(d_a, a, sizeof(float)*m*m, hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(j_error));
    hipMemcpy(d_b, b, sizeof(float)*m*m, hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 5: %s\n", hipGetErrorString(j_error));
    
    float (*f)(float*,float*,int,int,int) =  (float (*)(float*,float*,int,int,int)) get_anonymous_9nl89mhko6_ptr();
    

    map2xy2D_kernel<<<dimGrid, dimBlock>>>(d_a, d_b, m, d_c, m,f);  
   
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 6: %s\n", hipGetErrorString(j_error));
    
    hipDeviceSynchronize();
     j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Synchronize: %s\n", hipGetErrorString(j_error));

    hipMemcpy(c, d_c, sizeof(float)*m*m, hipMemcpyDeviceToHost);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(j_error));

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("cuda\t%d\t%3.1f\n", m,time);

    //clock_gettime(CLOCK_MONOTONIC_RAW, &end);

    //printf ("cuda   %d   %f \n",m,
     //       ((end.tv_nsec - begin.tv_nsec) / 1000000000.0 +
       //     (end.tv_sec  - begin.tv_sec))*1000);
   
//    cpu_mm(a,b,cpu_result,m,m,m);
  
  //  checkElementsAre(c,cpu_result,m*m);

    
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
    