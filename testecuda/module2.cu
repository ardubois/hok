
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(val) \
    cudaErrorCheck(val, __FILE__, __LINE__, true)
void cudaErrorCheck(hipError_t err, char* file, int line, bool abort)
{
    if(err != hipSuccess)
    {
        printf("%s %s %d\n", hipGetErrorString(err), file, line);
        if(abort) exit(-1);
    }
}

typedef float (*pfunc)(float arg);

__device__ float dev_func(float arg) {
    return arg * arg;
}

// create device function pointer here
__device__ pfunc dev_func_ptr = dev_func;

__global__ void ker_func(pfunc fnc) {
    // call function through device function pointer
    printf("%f\n", fnc(2));
}

extern "C" pfunc get_pointer()
{
    pfunc host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(dev_func_ptr), sizeof(pfunc)));
    return host_function_ptr;
   
}


extern "C" void launch(pfunc myptr)
{
    // create a host function pointer
 //   pfunc host_function_ptr;
    // copy function pointer value from device to host
   // gpuErrchk(cudaMemcpyFromSymbol(&host_function_ptr, dev_func_ptr, sizeof(pfunc)));
    // pass the copied function pointer in kernel
   // printf("my pointer %p\n", myptr);
   // printf("pointeiro %p\n", host_function_ptr);

    ker_func<<<1,1>>>(myptr);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

 
}