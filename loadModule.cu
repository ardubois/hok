#include "hip/hip_runtime.h"
__global__
void inc(float v)
{
 return v+1;
}

__global__
void inc_vet(float *result, float *a, int n)
{
	int i= (threadIdx.x + (blockIdx.x * blockDim.x));
	if(i < n)   
            result[i] = inc(a[i]);
}




int main (int argc, char *argv[]) {
	float *a, *resp, *dev_a, *dev_resp;

	int n = 10000;

	a = (float*)malloc(n*sizeof(float));

	
	resp = (float*)malloc(n*sizeof(float));

	for(int i=0; i<n; i++) {
		a[i] = i;
		
	}
    
	hipMalloc((void**)&dev_a, n*sizeof(float));
	hipMalloc((void**)&dev_resp, n*sizeof(float));
	
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
	
	inc_vet<<<blocksPerGrid, threadsPerBlock>>>(dev_result, dev_a , n);

	hipMemcpy(resp,dev_resp, n*sizeof(float), hipMemcpyDeviceToHost);

	c = 0;
	for(int i=0; i<10; i++) {
		printf("resp[%i] = %f", i,resp[i]);
	}
   
	hipFree(dev_a);
	hipFree(dev_resp;
    
	
	free(a);
	free(resp);
  	
}