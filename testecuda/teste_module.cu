

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <dlfcn.h>
typedef float (*func)(float);

int main()
{

printf("inicio.\n");
 void * m_handle = dlopen("module.so", RTLD_NOW);
  if(m_handle== NULL)  
      { char message[200];
        strcpy(message,"Error opening dll!! ");
      }


printf("m handle %p\n",m_handle);
char *errstr;


errstr = dlerror();
if (errstr != NULL)
printf ("A dynamic linking error occurred: (%s)\n", errstr);
//func (*fun)();
//fun= (func(*)())dlsym( m_handle, "inc_ptr");

//printf("ok1!\n");
//func pointer = fun();

//printf("ok2!\n");

void (*launch)();
launch= (void(*)())dlsym( m_handle, "launch");
printf("ok.\n");
if (launch==NULL) {printf("NULL\n");}
(*launch)();

}