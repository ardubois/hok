

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <dlfcn.h>
typedef float (*func)(float);

int main()
{

 void * m_handle = dlopen("./module2.so", RTLD_NOW);
  if (!m_handle) { 
   fprintf(stderr, "dlopen failure: %s\n", dlerror()); 
   exit (EXIT_FAILURE); }
char *errstr;
errstr = dlerror();
if (errstr != NULL)
printf ("A dynamic linking error occurred: (%s)\n", errstr);


func host_function_ptr;
func (*get_ptr)();
get_ptr = (func (*)())dlsym( m_handle, "get_pointer");
host_function_ptr = get_ptr();
printf("host function pointer main %p", host_function_ptr);


 void * m_handle2 = dlopen("./module2.so", RTLD_NOW);
  if (!m_handle2) { 
   fprintf(stderr, "dlopen failure: %s\n", dlerror()); 
   exit (EXIT_FAILURE); }
//char *errstr;
errstr = dlerror();
if (errstr != NULL)
printf ("A dynamic linking error occurred: (%s)\n", errstr);



void (*launch)(func);
launch= (void(*)(func))dlsym( m_handle2, "launch");
printf("ok.\n");
if (launch==NULL) {printf("NULL\n");}
(*launch)(host_function_ptr);

}