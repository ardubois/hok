
#include <hip/hip_runtime.h>
#include <stdio.h>
#define gpuErrchk(val) \
    cudaErrorCheck(val, __FILE__, __LINE__, true)

void cudaErrorCheck(hipError_t err, char* file, int line, bool abort)
{
    if(err != hipSuccess)
    {
        printf("%s %s %d\n", hipGetErrorString(err), file, line);
        if(abort) exit(-1);
    }
}

typedef float (*pfunc)(float arg);

__device__ float five_times(float arg) {
    return 5 * arg;
}

// create device function pointer here
__device__ pfunc five_times_ptr = five_times;

extern "C" pfunc get_ptr_five_times()
{
    pfunc host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(five_times_ptr), sizeof(pfunc)));
    return host_function_ptr;
   
}

