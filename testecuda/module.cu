#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <dlfcn.h>
__device__
float inc(float v)
{
 return v+2;
}

typedef float (*func)(float);

__device__ func ptr_inc_fun = inc;


extern "C" func ptr_inc()
{

    func pfun = NULL;
    hipMemcpy((void*)pfun,(void*)inc, sizeof(float(*)(float)), hipMemcpyDeviceToHost);
    return pfun;
}


__global__
void inc_vet(float *result, float *a, int n, float (*fun)(float))
{
	// void **fun_res;

	//fun = inc;
	int i= (threadIdx.x + (blockIdx.x * blockDim.x));
	if(i < n)   
            result[i] = fun(a[i]);
}

extern "C" void launch()
{
    printf("hello world\n");
    float *a, *resp, *dev_a, *dev_resp;


	int n = 10000;

	int block_size = 32;
	int nBlocks = (n + block_size - 1) / block_size;

	a = (float*)malloc(n*sizeof(float));
	resp = (float*)malloc(n*sizeof(float));

	for(int i=0; i<n; i++) {
		a[i] = i;
	}
    
	hipMalloc((void**)&dev_a, n*sizeof(float));
	hipMalloc((void**)&dev_resp, n*sizeof(float));
	
    hipMemcpy(dev_a, a, n*sizeof(float), hipMemcpyHostToDevice);

   float(*fptr)(float) =  NULL;

    hipMemcpy((void*)fptr,(void*)ptr_inc_fun, sizeof(float(*)(float)), hipMemcpyDeviceToHost);
     printf("cuda mem depois \n");
    inc_vet<<<nBlocks, block_size>>>(dev_resp, dev_a , n,fptr);

    hipError_t error_gpu = hipGetLastError();
    if(error_gpu != hipSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, hipGetErrorString(error_gpu));
       printf(message);
     }

	hipMemcpy(resp,dev_resp, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<10; i++) {
		printf("resp[%i] = %f\n", i,resp[i]);
	}
   
	hipFree(dev_a);
	hipFree(dev_resp);
    
	
	free(a);
	free(resp);


}