#define gpuErrchk(val) \
    cudaErrorCheck(val, __FILE__, __LINE__, true)
    
typedef float (*pfunc)(float arg);

__device__ float five_times(float arg) {
    return 5 * arg;
}

// create device function pointer here
__device__ pfunc five_times_ptr = five_times;

extern "C" pfunc get_ptr_five_times()
{
    pfunc host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(five_times_ptr), sizeof(pfunc)));
    return host_function_ptr;
   
}

