
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(val) \
    cudaErrorCheck(val, __FILE__, __LINE__, true)
void cudaErrorCheck(hipError_t err, char* file, int line, bool abort)
{
    if(err != hipSuccess)
    {
        printf("%s %s %d\n", hipGetErrorString(err), file, line);
        if(abort) exit(-1);
    }
}

typedef float (*pfunc)(float arg);

__device__ float dev_func(float arg) {
    return arg * arg;
}

// create device function pointer here
__device__ pfunc dev_func_ptr = dev_func;

__global__ void ker_func(pfunc fnc,pfunc func2) {
    // call function through device function pointer
    printf("%f\n", func2(fnc(2)));
}

extern "C" pfunc get_pointer()
{
    pfunc host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(dev_func_ptr), sizeof(pfunc)));
    return host_function_ptr;
   
}


__device__ float five_times(float arg) {
    return 5 * arg;
}

// create device function pointer here
__device__ pfunc five_times_ptr = five_times;

extern "C" pfunc get_ptr_five_times()
{
    pfunc host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(five_times_ptr), sizeof(pfunc)));
    return host_function_ptr;
   
}

extern "C"
__global__ void simple_kernel() {
    // call function through device function pointer
    printf("hello world");
}



extern "C" void launch(pfunc myptr,pfunc myptr2)
{
    // create a host function pointer
 //   pfunc host_function_ptr;
    // copy function pointer value from device to host
   // gpuErrchk(cudaMemcpyFromSymbol(&host_function_ptr, dev_func_ptr, sizeof(pfunc)));
    // pass the copied function pointer in kernel
   // printf("my pointer %p\n", myptr);
   // printf("pointeiro %p\n", host_function_ptr);

    ker_func<<<1,1>>>(myptr,myptr2);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

 
}