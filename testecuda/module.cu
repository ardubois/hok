#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <dlfcn.h>

__device__
float inc(float v)
{
 return v+2;
}

typedef float (*func)(float);

__device__ func ptr_inc_fun = inc;


extern "C" func ptr_inc()
{

    func pfun = NULL;
    hipMemcpy((void*)pfun,(void*)inc, sizeof(float(*)(float)), hipMemcpyDeviceToHost);
    return pfun;
}


__global__
void inc_vet(float *result, float *a, int n, float (*fun)(float))
{
	// void **fun_res;
    printf("fun %p\n",fun);
	printf("inc %p\n",inc);
	fun = ptr_inc_fun;
	int i= (threadIdx.x + (blockIdx.x * blockDim.x));
	if(i < n)   
            result[i] = fun(a[i]);
}

__global__
void copy_ptr(func f)
{
	 f = ptr_inc_fun;
	 printf("kernel f %p\n", f);
}

extern "C" void launch()
{
    printf("hello world\n");
    float *a, *resp, *dev_a, *dev_resp;


	int n = 10000;

	int block_size = 32;
	int nBlocks = (n + block_size - 1) / block_size;

	a = (float*)malloc(n*sizeof(float));
	resp = (float*)malloc(n*sizeof(float));

	for(int i=0; i<n; i++) {
		a[i] = i;
	}
    

	func dev_inc_pointer;
	func host_inc_pointer;

    printf("before cuda malloc\n");
	hipMalloc((void**) &dev_inc_pointer, sizeof(func));
     printf("after cuda malloc\n");
	copy_ptr<<<1,1>>>(dev_inc_pointer);
   printf("afterkernel launch %p\n", host_inc_pointer);
	hipMemcpyFromSymbol((void*) host_inc_pointer, (void*) dev_inc_pointer, sizeof(func), hipMemcpyDeviceToHost);
    printf("after memory  copy %p\n",host_inc_pointer);
//	printf("pointer copied %p\n", host_inc_pointer);


	hipMalloc((void**)&dev_a, n*sizeof(float));
	hipMalloc((void**)&dev_resp, n*sizeof(float));
	
    hipMemcpy(dev_a, a, n*sizeof(float), hipMemcpyHostToDevice);

   float(*fptr)(float) =  NULL;

    hipMemcpy((void*)fptr,(void*) inc, sizeof(float(*)(float)), hipMemcpyDeviceToHost);
     printf("cuda mem depois \n %f", fptr);
   // inc_vet<<<nBlocks, block_size>>>(dev_resp, dev_a , n,ptr_inc_fun);
inc_vet<<<2, 2>>>(dev_resp, dev_a , n,fptr);
    hipError_t error_gpu = hipGetLastError();
    if(error_gpu != hipSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, hipGetErrorString(error_gpu));
       printf(message);
     }

	hipMemcpy(resp,dev_resp, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<10; i++) {
		printf("resp[%i] = %f\n", i,resp[i]);
	}
   
	hipFree(dev_a);
	hipFree(dev_resp);
    
	
	free(a);
	free(resp);


}