
#include <hip/hip_runtime.h>


__device__ static float atomic_cas(float* address, float oldv, float newv)
{
    int* address_as_i = (int*) address;
    return  __int_as_float(atomicCAS(address_as_i, __float_as_int(oldv), __float_as_int(newv)));
}


