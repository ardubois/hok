#include "hip/hip_runtime.h"
#include "erl_nif.h"


__device__
float sum(float a, float b)
{
return((a + b));
}

__device__ void* sum_ptr = (void*) sum;

extern "C" void* get_sum_ptr()
{
	void* host_function_ptr;
	hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(sum_ptr), sizeof(void*));
	return host_function_ptr;
}



__global__
void map2(float *a1, float *a2, float *a3, int size, float (*f)(float,float))
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
if((id < size))
{
	a3[id] = (*f)(a1[id], a2[id]);
}

}

extern "C" void map2_call(ErlNifEnv *env, const ERL_NIF_TERM argv[], ErlNifResourceType* type,ErlNifResourceType* ftype)
  {

    ERL_NIF_TERM list;
    ERL_NIF_TERM head;
    ERL_NIF_TERM tail;
    float **array_res;
    void **fun_res;

    const ERL_NIF_TERM *tuple_blocks;
    const ERL_NIF_TERM *tuple_threads;
    int arity;

    if (!enif_get_tuple(env, argv[1], &arity, &tuple_blocks)) {
      printf ("spawn: blocks argument is not a tuple");
    }

    if (!enif_get_tuple(env, argv[2], &arity, &tuple_threads)) {
      printf ("spawn:threads argument is not a tuple");
    }
    int b1,b2,b3,t1,t2,t3;

    enif_get_int(env,tuple_blocks[0],&b1);
    enif_get_int(env,tuple_blocks[1],&b2);
    enif_get_int(env,tuple_blocks[2],&b3);
    enif_get_int(env,tuple_threads[0],&t1);
    enif_get_int(env,tuple_threads[1],&t2);
    enif_get_int(env,tuple_threads[2],&t3);

    dim3 blocks(b1,b2,b3);
    dim3 threads(t1,t2,t3);

    list= argv[3];

  enif_get_list_cell(env,list,&head,&tail);
  float *arg1 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  float *arg2 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  float *arg3 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  int arg4;
  enif_get_int(env, head, &arg4);
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
      float (*arg5)(float,float) = (float (*)(float,float))*fun_res;
      list = tail;

       map2<<<blocks, threads>>>(arg1,arg2,arg3,arg4,arg5);
    hipError_t error_gpu = hipGetLastError();
    if(error_gpu != hipSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, hipGetErrorString(error_gpu));
       enif_raise_exception(env,enif_make_string(env, message, ERL_NIF_LATIN1));
     }
}

__device__
float anonymous_dm1n5andmc(float x, float y)
{


	float n = (x + y);
return(n);
}

__device__ void* anonymous_dm1n5andmc_ptr = (void*) anonymous_dm1n5andmc;

extern "C" void* get_anonymous_dm1n5andmc_ptr()
{
	void* host_function_ptr;
	hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_dm1n5andmc_ptr), sizeof(void*));
	return host_function_ptr;
}



__device__
float anonymous_gg5n77jemg(float x, float y)
{


	float n = (x + y);
return(n);
}

__device__ void* anonymous_gg5n77jemg_ptr = (void*) anonymous_gg5n77jemg;

extern "C" void* get_anonymous_gg5n77jemg_ptr()
{
	void* host_function_ptr;
	hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_gg5n77jemg_ptr), sizeof(void*));
	return host_function_ptr;
}



__device__
float anonymous_hna3kgf5ng(float x, float y)
{


	float n = (x + y);
return(n);
}

__device__ void* anonymous_hna3kgf5ng_ptr = (void*) anonymous_hna3kgf5ng;

extern "C" void* get_anonymous_hna3kgf5ng_ptr()
{
	void* host_function_ptr;
	hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_hna3kgf5ng_ptr), sizeof(void*));
	return host_function_ptr;
}



__device__
float anonymous_6c7bn66210(float x, float y)
{


	float n = (x + y);
return(n);
}

__device__ void* anonymous_6c7bn66210_ptr = (void*) anonymous_6c7bn66210;

extern "C" void* get_anonymous_6c7bn66210_ptr()
{
	void* host_function_ptr;
	hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_6c7bn66210_ptr), sizeof(void*));
	return host_function_ptr;
}


