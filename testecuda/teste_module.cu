

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <dlfcn.h>
typedef float (*func)(float);

int main()
{

 void * m_handle = dlopen("./module.so", RTLD_NOW);
  if (!m_handle) { 
   fprintf(stderr, "dlopen failure: %s\n", dlerror()); 
   exit (EXIT_FAILURE); }
char *errstr;
errstr = dlerror();
if (errstr != NULL)
printf ("A dynamic linking error occurred: (%s)\n", errstr);


func host_function_ptr;
func (*get_ptr)();
get_ptr = (func (*)())dlsym( m_handle, "get_ptr_five_times");
host_function_ptr = get_ptr();
printf("host function pointer main %p", host_function_ptr);


func host_function_ptr2;
func (*get_ptr2)();
get_ptr2 = (func (*)())dlsym( m_handle, "get_pointer");
host_function_ptr2 = get_ptr2();
printf("host function pointer main %p", host_function_ptr2);


/* void * m_handle2 = dlopen("./module.so", RTLD_NOW);
  if (!m_handle2) { 
   fprintf(stderr, "dlopen failure: %s\n", dlerror()); 
   exit (EXIT_FAILURE); }
//char *errstr;
errstr = dlerror();
if (errstr != NULL)
printf ("A dynamic linking error occurred: (%s)\n", errstr);

*/

void (*launch)(func,func);
launch= (void(*)(func,func))dlsym( m_handle, "launch");
printf("ok.\n");
if (launch==NULL) {printf("NULL\n");}
(*launch)(host_function_ptr,host_function_ptr2);

}