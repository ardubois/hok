#include "hip/hip_runtime.h"
#include<stdio.h>


#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <hip/hip_runtime.h>


extern "C"
__device__
float inc(float v)
{
 return v+2;
}

__global__
void inc_vet(float *result, float *a, int n,float (*fun)(float))
{
	// void **fun_res;

	fun = inc;
	int i= (threadIdx.x + (blockIdx.x * blockDim.x));
	if(i < n)   
            result[i] = fun(a[i]);
}

__device__ float (*fun_pointer)(float) = inc;


int main (int argc, char *argv[]) {
	float *a, *resp, *dev_a, *dev_resp;

	float (*pfun)(float);

	int n = 10000;

	int block_size = 32;
	int nBlocks = (n + block_size - 1) / block_size;

	printf("block_size = %d   nBlocks = %d total = %d\n", block_size,nBlocks,block_size*nBlocks);

	a = (float*)malloc(n*sizeof(float));

	
	resp = (float*)malloc(n*sizeof(float));

	for(int i=0; i<n; i++) {
		a[i] = i;
		
	}
    
	hipMalloc((void**)&dev_a, n*sizeof(float));
	hipMalloc((void**)&dev_resp, n*sizeof(float));
	
    hipMemcpy(dev_a, a, n*sizeof(float), hipMemcpyHostToDevice);


	hipModule_t cuModule;

    int ret = hipModuleLoad(&cuModule, "/home/dubois/hok/loadModule.ptx");

	printf("retorno %d\n", ret);

	hipFunction_t function;
    int funs = 0;
//	ret = hipModuleGetFunctionCount(&funs,cuModule) ;

	printf("retorno %d funs= %d\n", ret,funs);


	ret = hipModuleGetFunction(&function, cuModule, "inc");

	printf("retorno %d\n", ret);

    hipMemcpy((void*)pfun,(void*)inc, sizeof(float(*)(float)), hipMemcpyDeviceToHost);


	inc_vet<<<nBlocks, block_size>>>(dev_resp, dev_a , n,pfun);

	hipMemcpy(resp,dev_resp, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<10; i++) {
		printf("resp[%i] = %f\n", i,resp[i]);
	}
   
	hipFree(dev_a);
	hipFree(dev_resp);
    
	
	free(a);
	free(resp);
  	
}