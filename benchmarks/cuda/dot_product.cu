
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>



__device__ static float atomic_cas(float* address, float oldv, float newv)
{
    int* address_as_i = (int*) address;
    return  __int_as_float(atomicCAS(address_as_i, __float_as_int(oldv), __float_as_int(newv)));
}



__global__
void map_2kernel(float *a1, float *a2, float *a3, int size, float (*f)(float,float))
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
if((id < size))
{
        a3[id] = f(a1[id], a2[id]);
}

}

__global__
void reduce_kernel(float *a, float *ref4, float (*f)(float,float), int n)
{
__shared__ float cache[256];
        int tid = (threadIdx.x + (blockIdx.x * blockDim.x));
        int cacheIndex = threadIdx.x;
        float temp = 0.0;
while((tid < n)){
        temp = f(a[tid], temp);
        tid = ((blockDim.x * gridDim.x) + tid);
}
        cache[cacheIndex] = temp;
__syncthreads();
        int i = (blockDim.x / 2);
while((i != 0)){
if((cacheIndex < i))
{
        cache[cacheIndex] = f(cache[(cacheIndex + i)], cache[cacheIndex]);
}

__syncthreads();
        i = (i / 2);
}
if((cacheIndex == 0))
{
        float current_value = ref4[0];
while((! (current_value == atomic_cas(ref4, current_value, f(cache[0], current_value))))){
        current_value = ref4[0];
}
}

}

//#############################


__device__
float anonymous_mult(float a, float b)
{
return ((a * b));
}

__device__ void* anonymous_mult_ptr = (void*) anonymous_mult;

extern "C" void* get_anonymous_mult_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_mult_ptr), sizeof(void*));
        return host_function_ptr;
}


//#############################


__device__
float anonymous_sum(float a, float b)
{
return ((a + b));
}

__device__ void* anonymous_sum_ptr = (void*) anonymous_sum;

extern "C" void* get_anonymous_sum_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(anonymous_sum_ptr), sizeof(void*));
        return host_function_ptr;
}




int main(int argc, char *argv[])
{

    float *a, *b, *resp;
	float *dev_a, *dev_b, *dev_resp;
    hipError_t j_error;

    int N = atoi(argv[1]);

    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    resp = (float*)malloc(N*sizeof(float));

    for(int i=0; i<N; i++) {
		a[i] = rand();
		
	}

    for(int i=0; i<N; i++) {
		b[i] = rand();
		
	}

    int threadsPerBlock = 256;
    int  numberOfBlocks = (N + threadsPerBlock - 1)/ threadsPerBlock;

    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


	hipMalloc((void**)&dev_a, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMalloc((void**)&dev_b, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMalloc((void**)&dev_resp, N*sizeof(float));
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    float (*f1)(float,float) = (float (*)(float,float)) get_anonymous_mult_ptr();
    float (*f2)(float,float) = (float (*)(float,float)) get_anonymous_sum_ptr();

    float *final, *d_final;
    final = (float *)malloc(sizeof(float));
	hipMalloc((void **) &d_final,sizeof(float));

    map_2kernel<<< numberOfBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_resp, N, f1);
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    reduce_kernel<<< numberOfBlocks, threadsPerBlock>>>(dev_resp, d_final, f2, N);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

     hipMemcpy( final, d_final, sizeof(float), hipMemcpyDeviceToHost );
     j_error = hipGetLastError();
    if(j_error != hipSuccess) {printf("Error: %s\n", hipGetErrorString(j_error)); exit(1);}

    
    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_resp);
    hipFree(d_final);
    
	hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", N,time);

/*
    for(int i=0; i<10; i++) {
		printf("resp[%d] = %f;\n",i,resp[i]);
	}

*/
	//printf("\n FINAL RESULTADO: %f \n", c);

	free(a);
    free(b);
	free(resp);
    free(final);

}
