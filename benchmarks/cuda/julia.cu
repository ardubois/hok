
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <malloc.h>

#define _bitsperpixel 32
#define _planes 1
#define _compression 0
#define _xpixelpermeter 0x13B //0x130B //2835 , 72 DPI
#define _ypixelpermeter 0x13B//0x130B //2835 , 72 DPI

#pragma pack(push,1)
typedef struct{
    uint8_t signature[2];
    uint32_t filesize;
    uint32_t reserved;
    uint32_t fileoffset_to_pixelarray;
} fileheader;
typedef struct{
    uint32_t dibheadersize;
    uint32_t width;
    uint32_t height;
    uint16_t planes;
    uint16_t bitsperpixel;
    uint32_t compression;
    uint32_t imagesize;
    uint32_t ypixelpermeter;
    uint32_t xpixelpermeter;
    uint32_t numcolorspallette;
    uint32_t mostimpcolor;
} bitmapinfoheader;
typedef struct {
    fileheader fileheader;
    bitmapinfoheader bitmapinfoheader;
} bitmap;
#pragma pack(pop)

void genBpm (int height, int width, float *pixelbuffer_f) {
    uint32_t pixelbytesize = height*width*_bitsperpixel/8;
    uint32_t  _filesize =pixelbytesize+sizeof(bitmap);
    FILE *fp = fopen("julia.bmp","wb");
    bitmap *pbitmap  = (bitmap*)calloc(1,sizeof(bitmap));

    int buffer_size = height*width*4;
    uint8_t *pixelbuffer = (uint8_t*)malloc(buffer_size);

    for(int i = 0; i<buffer_size;i++)
    {
     pixelbuffer[i]= (uint8_t) pixelbuffer_f[i];
    }


    //strcpy(pbitmap->fileheader.signature,"BM");
    pbitmap->fileheader.signature[0] = 'B';
    pbitmap->fileheader.signature[1] = 'M';
    pbitmap->fileheader.filesize = _filesize;
    pbitmap->fileheader.fileoffset_to_pixelarray = sizeof(bitmap);
    pbitmap->bitmapinfoheader.dibheadersize =sizeof(bitmapinfoheader);
    pbitmap->bitmapinfoheader.width = width;
    pbitmap->bitmapinfoheader.height = height;
    pbitmap->bitmapinfoheader.planes = _planes;
    pbitmap->bitmapinfoheader.bitsperpixel = _bitsperpixel;
    pbitmap->bitmapinfoheader.compression = _compression;
    pbitmap->bitmapinfoheader.imagesize = pixelbytesize;
    pbitmap->bitmapinfoheader.ypixelpermeter = _ypixelpermeter ;
    pbitmap->bitmapinfoheader.xpixelpermeter = _xpixelpermeter ;
    pbitmap->bitmapinfoheader.numcolorspallette = 0;
    fwrite (pbitmap, 1, sizeof(bitmap),fp);
    //memset(pixelbuffer,pixel,pixelbytesize);
    fwrite(pixelbuffer,1,pixelbytesize,fp);
    fclose(fp);
    free(pbitmap);
    free(pixelbuffer);
}


__device__
int julia(int x, int y, int dim)
{
float scale = 0.1;
float jx = ((scale * (dim - x)) / dim);
float jy = ((scale * (dim - y)) / dim);
float cr = (- 0.8);
float ci = 0.156;
float ar = jx;
float ai = jy;
for( int i = 0; i<200; i++){
float nar = (((ar * ar) - (ai * ai)) + cr);
float nai = (((ai * ar) + (ar * ai)) + ci);
if((((nar * nar) + (nai * nai)) > 1.0e3))
{
return (0);
}

        ar = nar;
        ai = nai;
}

return (1);
}



__device__ void* julia_ptr = (void*) julia;

extern "C" void* get_julia_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(julia_ptr), sizeof(void*));
        return host_function_ptr;
}



__device__
int julia_function(float *ptr, int x, int y, int dim)
{
int offset = (x + (y * dim));
float juliaValue = julia(x, y, dim);
        ptr[((offset * 4) + 0)] = (255 * juliaValue);
        ptr[((offset * 4) + 1)] = 0;
        ptr[((offset * 4) + 2)] = 0;
        ptr[((offset * 4) + 3)] = 255;
return (1);
}

__device__ void* julia_function_ptr = (void*) julia_function;

extern "C" void* get_julia_function_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(julia_function_ptr), sizeof(void*));
        return host_function_ptr;
}



__global__
void mapgen2D_xy_1para_noret_ker(float *resp, int arg1, int size, int (*f)(float*,int,int,int))
{
int x = ((blockIdx.x * blockDim.x) + threadIdx.x);
int y = ((blockIdx.y * blockDim.y) + threadIdx.y);
if(((x < size) && (y < size)))
{
int v = f(resp, x, y, arg1);
}

}

int main( int argc, char const *argv[] ) {

    int usr_value = atoi(argv[1]);
   
    int height = usr_value;
    int width  = usr_value;
    int DIM = usr_value;
    int size_array = height*width*4*sizeof(float);
    hipError_t j_error;
    
    //int pixelbytesize=  height*width*_bitsperpixel/8;
    //printf(" pixel byte size %lu\n",pixelbytesize);
   
    float time;
    hipEvent_t start, stop;   
     hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


     float *h_pixelbuffer = (float*)malloc(size_array);
     float *d_pixelbuffer;

     ////////
    hipMalloc( (void**)&d_pixelbuffer, size_array);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(j_error));
    ////////

    
    ////////////////////
    dim3 grid(DIM,DIM);

    int (*f)(float*,int,int,int) = (int (*)(float*,int,int,int)) get_julia_function_ptr();

    mapgen2D_xy_1para_noret_ker<<<grid, 1>>>(d_pixelbuffer,DIM,DIM,f);
    
  j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(j_error));
  ////////



    hipMemcpy(h_pixelbuffer, d_pixelbuffer, size_array, hipMemcpyDeviceToHost); // return results 
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(j_error));



    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", usr_value,time);
    
    //genBpm(height,width,h_pixelbuffer);
   
    free(h_pixelbuffer);
    hipFree(d_pixelbuffer);
}



