
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(val) \
    cudaErrorCheck(val, __FILE__, __LINE__, true)
void cudaErrorCheck(hipError_t err, char* file, int line, bool abort)
{
    if(err != hipSuccess)
    {
        printf("%s %s %d\n", hipGetErrorString(err), file, line);
        if(abort) exit(-1);
    }
}

typedef float (*pfunc)(float arg);

__device__ float dev_func(float arg) {
    return arg * arg;
}
// create device function pointer here
__device__ void* dev_func_ptr = (void*) dev_func;

extern "C" void* get_pointer()
{
    void* host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(dev_func_ptr), sizeof(void*)));
    return host_function_ptr;
   
}


__device__ float five_times(float arg) {
    return 5 * arg;
}

// create device function pointer here
__device__ void* five_times_ptr = (void*) five_times;

extern "C" void* get_ptr_five_times()
{
    void* host_function_ptr;
    // copy function pointer value from device to host
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(five_times_ptr), sizeof(void*)));
    return host_function_ptr;
   
}


__global__ void ker_func(pfunc fnc,pfunc func2) {
    // call function through device function pointer
    printf("%f\n", func2(fnc(2)));
}


extern "C" void launch(pfunc myptr,pfunc myptr2)
{
    // create a host function pointer
 //   pfunc host_function_ptr;
    // copy function pointer value from device to host
   // gpuErrchk(cudaMemcpyFromSymbol(&host_function_ptr, dev_func_ptr, sizeof(pfunc)));
    // pass the copied function pointer in kernel
   // printf("my pointer %p\n", myptr);
   // printf("pointeiro %p\n", host_function_ptr);

    ker_func<<<1,1>>>(myptr,myptr2);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

 
}