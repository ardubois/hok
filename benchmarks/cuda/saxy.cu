
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__device__
float saxpy(float a, float b)
{
return (((2 * a) + b));
}

__device__ void* saxpy_ptr = (void*) saxpy;

extern "C" void* get_saxpy_ptr()
{
        void* host_function_ptr;
        hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(saxpy_ptr), sizeof(void*));
        return host_function_ptr;
}

__global__
void map_2kernel(float *a1, float *a2, float *a3, int size, float (*f)(float,float))
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
if((id < size))
{
        a3[id] = f(a1[id], a2[id]);
}

}

int main(int argc, char *argv[])
{

    float *a, *b, *resp;
	float *dev_a, *dev_b, *dev_resp;

    int N = atoi(argv[1]);

    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    resp = (float*)malloc(N*sizeof(float));

    for(int i=0; i<N; i++) {
		a[i] = i+1;
		
	}

    for(int i=0; i<N; i++) {
		b[i] = i+1;
		
	}


    int threadsPerBlock = 128;
    int  numberOfBlocks = (N + threadsPerBlock - 1)/ threadsPerBlock;

   

    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


	hipMalloc((void**)&dev_a, N*sizeof(float));
	hipMalloc((void**)&dev_b, N*sizeof(float));
	hipMalloc((void**)&dev_resp, N*sizeof(float));
	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    float (*f)(float,float) = (float (*)(float,float)) get_saxpy_ptr();

    map_2kernel<<< numberOfBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_resp, N, f);

    hipMemcpy(resp, dev_resp, N*sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<10; i++) {
		printf("resp[%d] = %f;\n",i,resp[i]);
	}

    hipFree(dev_a);
	hipFree(dev_b);
    hipFree(dev_resp);
    
	hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", N,time);

	//printf("\n FINAL RESULTADO: %f \n", c);

	free(a);
    free(b);
	free(resp);

}
