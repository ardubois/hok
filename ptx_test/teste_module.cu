
#include <stdint.h>
#include <stdio.h>
#include <dlfcn.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr,
                "CUDA Driver API error = %04d from file <%s>, line %i.\n",
                err, file, line );
        exit(-1);
    }
}  


typedef float (*func)(float);

int main()
{

hipDevice_t   device;
hipCtx_t  context;
hipModule_t   module;
hipFunction_t function;

hipError_t err = hipInit(0);

checkCudaErrors(hipDeviceGet(&device, 0));

err = hipCtxCreate(&context, 0, device);
if (err != hipSuccess) {
        fprintf(stderr, "* Error initializing the CUDA context.\n");
        cuCtxDetach(context);
        exit(-1);
}

err = hipModuleLoad(&module, "module.ptx");
if (err != hipSuccess) {
        fprintf(stderr, "* Error loading the module %s\n", "module.ptx");
        cuCtxDetach(context);
        exit(-1);
}

err = hipModuleGetFunction(&function, module, "simple_kernel");

if (err != hipSuccess) {
        fprintf(stderr, "* Error getting kernel function %s\n", "simple_kernel");
        cuCtxDetach(context);
        exit(-1);
}

checkCudaErrors( hipModuleLaunchKernel(function, 1, 1, 1,  // Nx1x1 blocks
                                    1, 1, 1,            // 1x1x1 threads
                                    0, 0, {}, 0) );


}